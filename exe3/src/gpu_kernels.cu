#include "hip/hip_runtime.h"
/*
 *  gpu_kernels.cu -- GPU kernels
 *
 *  Copyright (C) 2014, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2014, Vasileios Karakasis
 */ 

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "error.h"
#include "gpu_util.h"
#include "graph.h"
#include "timer.h"

#define GPU_KERNEL_NAME(name)   do_apsp_gpu ## name

weight_t *copy_graph_to_gpu(const graph_t *graph)
{
    size_t dist_size = graph->nr_vertices*graph->nr_vertices;
    weight_t *dist_gpu = (weight_t *) gpu_alloc(dist_size*sizeof(*dist_gpu));
    if (!dist_gpu)
        error(0, "gpu_alloc() failed: %s", gpu_get_last_errmsg());

    if (copy_to_gpu(graph->weights[0], dist_gpu,
                    dist_size*sizeof(*dist_gpu)) < 0)
        error(0, "copy_to_gpu() failed: %s", gpu_get_last_errmsg());

    return dist_gpu;
}

graph_t *copy_graph_from_gpu(const weight_t *dist_gpu, graph_t *graph)
{
    size_t dist_size = graph->nr_vertices*graph->nr_vertices;

    if (copy_from_gpu(graph->weights[0], dist_gpu,
                      dist_size*sizeof(*dist_gpu)) < 0)
        error(0, "copy_from_gpu() failed: %s", gpu_get_last_errmsg());

    return graph;
}

/*
 *  The naive GPU kernel
 */ 
__global__ void GPU_KERNEL_NAME(_naive)(weight_t *dist, int n, int k)
{
    int tid = (blockDim.x*blockDim.y*blockIdx.x)+ // consider line grid
        (threadIdx.x*blockDim.y + threadIdx.y);   // and global order 

    if (tid > n*n)
        return;

    int row = tid / n;
    int col = tid % n;

    dist[tid] = MIN(dist[tid], dist[row*n+k]+dist[k*n+col]);  
}

/*
 *  The tiled GPU kernel(s) using global memory
 */ 
__global__ void GPU_KERNEL_NAME(_tiled_stage_1)(weight_t *dist, int n,
        int k_tile)
{
    weight_t * a;
    weight_t * b;
    weight_t * c;
    /*single block*/
    a = &dist[k_tile*GPU_TILE_DIM+k_tile*GPU_TILE_DIM*n]; // Tkk
    b = a;
    c = a;

    int row = threadIdx.x;  // row-cal in the small square
    int col = threadIdx.y;
    
    for (int kk =0;kk<GPU_TILE_DIM;kk++) { 
        a[row*n+col] = MIN(a[row*n+col], b[row*n+kk]+c[kk*n+col]);
        __syncthreads();
    }
}


__global__ void GPU_KERNEL_NAME(_tiled_stage_2)(weight_t *dist, int n,
        int k_tile)
{

    weight_t * a;
    weight_t * b;
    weight_t * c;
    /*column grid*/
    if (blockIdx.y == k_tile)
        return;
    a = &dist[k_tile*GPU_TILE_DIM+blockIdx.y*GPU_TILE_DIM*n]; // Tik
    b = a;
    c = &dist[k_tile*GPU_TILE_DIM+k_tile*GPU_TILE_DIM*n]; //Tkk

    int row = threadIdx.x;  // row-cal in the small square
    int col = threadIdx.y;
    
    for (int kk=0;kk<GPU_TILE_DIM;kk++) {
        a[row*n+col] = MIN(a[row*n+col], b[row*n+kk]+c[kk*n+col]);
        __syncthreads();
    }
}

__global__ void GPU_KERNEL_NAME(_tiled_stage_3)(weight_t *dist, int n,
        int k_tile)
{
    weight_t * a;
    weight_t * b;
    weight_t * c;
    /*line grid*/
    if (blockIdx.x == k_tile)
        return;
    a = &dist[k_tile*GPU_TILE_DIM*n+blockIdx.x*GPU_TILE_DIM]; //Tki
    b = &dist[k_tile*GPU_TILE_DIM+k_tile*GPU_TILE_DIM*n]; //Tkk
    c = a;

    int row = threadIdx.x;  // row-cal in the small square
    int col = threadIdx.y;

    for (int kk=0;kk<GPU_TILE_DIM;kk++) {
        a[row*n+col] = MIN(a[row*n+col], b[row*n+kk]+c[kk*n+col]);
        __syncthreads();
    }
}
__global__ void GPU_KERNEL_NAME(_tiled_stage_4)(weight_t *dist, int n,
        int k_tile)
{
    weight_t * a;
    weight_t * b;
    weight_t * c;
    /*square grid*/
    if ((blockIdx.x == k_tile) || (blockIdx.y == k_tile))
        return;
    a = &dist[blockIdx.y*GPU_TILE_DIM*n+blockIdx.x*GPU_TILE_DIM]; //Tij
    b = &dist[blockIdx.y*n*GPU_TILE_DIM+k_tile*GPU_TILE_DIM]; //Tik
    c = &dist[k_tile*GPU_TILE_DIM*n+blockIdx.x*GPU_TILE_DIM]; //Tkj

    int row = threadIdx.x;  // row-cal in the small square
    int col = threadIdx.y;
    
    for (int kk=0;kk<GPU_TILE_DIM;kk++) {
        a[row*n+col] = MIN(a[row*n+col], b[row*n+kk]+c[kk*n+col]);
        __syncthreads();
    }
}

/*
 *  FILLME: Use different kernels for the different stages of the
 *  tiled FW computation
 *
 *  Use GPU_TILE_DIM (see graph.h) as the tile dimension. You can
 *  adjust its value during compilation. See `make help' for more
 *  information.
 */ 

/*
 *  The tiled GPU kernel(s) using shared memory
 */ 
__global__ void GPU_KERNEL_NAME(_tiled_shmem_stage_1)(weight_t *dist, int n,
                                                      int k_tile){

    int tid = threadIdx.x*blockDim.y + threadIdx.y;

    __shared__ weight_t  a [GPU_TILE_DIM * GPU_TILE_DIM];
    __shared__ weight_t  * b;
    __shared__ weight_t  * c;


    int row = tid / GPU_TILE_DIM;  // row-cal in the small square
    int col = tid % GPU_TILE_DIM;
    
    a[row*GPU_TILE_DIM + col] = dist[k_tile*GPU_TILE_DIM + k_tile*GPU_TILE_DIM*n + row*n + col];
    b=a;
    c=a; 
    __syncthreads();
 
    for (int kk =0;kk<GPU_TILE_DIM;kk++) { 
        a[row*GPU_TILE_DIM+col] = MIN(a[row*GPU_TILE_DIM+col], b[row*GPU_TILE_DIM+kk]+c[kk*GPU_TILE_DIM+col]);
        __syncthreads();
    }

    dist[k_tile*GPU_TILE_DIM + k_tile*GPU_TILE_DIM*n + row*n + col] = a[row*GPU_TILE_DIM + col] ;
   
}

__global__ void GPU_KERNEL_NAME(_tiled_shmem_stage_2)(weight_t *dist, int n, int k_tile){

    int tid = threadIdx.x*blockDim.y + threadIdx.y;

    __shared__ weight_t  a[GPU_TILE_DIM * GPU_TILE_DIM];
    __shared__ weight_t * b;
    __shared__ weight_t  c[GPU_TILE_DIM * GPU_TILE_DIM];

    /*column grid*/
    if (blockIdx.y == k_tile)
        return;

    int row = tid / GPU_TILE_DIM;  // row-cal in the small square
    int col = tid % GPU_TILE_DIM;

    a[row*GPU_TILE_DIM+col] = dist[k_tile*GPU_TILE_DIM + blockIdx.y*GPU_TILE_DIM*n + row*n + col];
    __syncthreads();
    b=a;
    c[row*GPU_TILE_DIM+col] = dist[k_tile*GPU_TILE_DIM + k_tile*GPU_TILE_DIM*n + row*n + col];
    __syncthreads();
    
    for (int kk=0;kk<GPU_TILE_DIM;kk++) {
        a[row*GPU_TILE_DIM+col] = MIN(a[row*GPU_TILE_DIM+col], b[row*GPU_TILE_DIM+kk]+c[kk*GPU_TILE_DIM+col]);
        __syncthreads();
    }
      
    dist[k_tile*GPU_TILE_DIM + blockIdx.y*GPU_TILE_DIM*n +row*n+col] = a[row*GPU_TILE_DIM + col] ;
    __syncthreads(); 
}

__global__ void GPU_KERNEL_NAME(_tiled_shmem_stage_3)(weight_t *dist, int n, int k_tile){

    int tid = threadIdx.x*blockDim.y + threadIdx.y;

    __shared__  weight_t  a[GPU_TILE_DIM*GPU_TILE_DIM] ;
    __shared__  weight_t  b[GPU_TILE_DIM*GPU_TILE_DIM];
    __shared__  weight_t * c;

    /*line grid*/
    if (blockIdx.x == k_tile)
        return;

    int row = tid / GPU_TILE_DIM;  // row-cal in the small square
    int col = tid % GPU_TILE_DIM;

    a[row*GPU_TILE_DIM +col] = dist[k_tile*GPU_TILE_DIM*n + blockIdx.x*GPU_TILE_DIM +row*n + col];
    __syncthreads();
    b[row*GPU_TILE_DIM +col]= dist[k_tile*GPU_TILE_DIM + k_tile*GPU_TILE_DIM*n + row*n +col];
    __syncthreads();
    c=a;
 
    for (int kk=0;kk<GPU_TILE_DIM;kk++) {
        a[row*GPU_TILE_DIM+col] = MIN(a[row*GPU_TILE_DIM+col], b[row*GPU_TILE_DIM+kk]+c[kk*GPU_TILE_DIM+col]);
        __syncthreads();
    }

    dist[k_tile*GPU_TILE_DIM*n + blockIdx.x*GPU_TILE_DIM +row*n + col] = a[row*GPU_TILE_DIM + col];
    __syncthreads(); 
}


__global__ void GPU_KERNEL_NAME(_tiled_shmem_stage_4)(weight_t *dist, int n, int k_tile){

    int tid = threadIdx.x*blockDim.y + threadIdx.y;

    __shared__  weight_t  a[GPU_TILE_DIM*GPU_TILE_DIM];
    __shared__  weight_t  b[GPU_TILE_DIM*GPU_TILE_DIM];
    __shared__  weight_t  c[GPU_TILE_DIM*GPU_TILE_DIM];

    /*square grid*/
    if ((blockIdx.x == k_tile) || (blockIdx.y == k_tile))
        return;
  
    int row = tid / GPU_TILE_DIM;  // row-cal in the small square
    int col = tid % GPU_TILE_DIM;

    a[row*GPU_TILE_DIM+col] = dist[blockIdx.y*GPU_TILE_DIM*n+blockIdx.x*GPU_TILE_DIM + row*n + col]; //Tij
    b[row*GPU_TILE_DIM+col] = dist[blockIdx.y*n*GPU_TILE_DIM+k_tile*GPU_TILE_DIM + row*n + col]; //Tik
    c[row*GPU_TILE_DIM+col] = dist[k_tile*GPU_TILE_DIM*n+blockIdx.x*GPU_TILE_DIM + row*n + col]; //Tkj
    __syncthreads();

    for (int kk=0;kk<GPU_TILE_DIM;kk++) {
        a[row*GPU_TILE_DIM+col] = MIN(a[row*GPU_TILE_DIM+col], b[row*GPU_TILE_DIM+kk]+c[kk*GPU_TILE_DIM+col]);
        __syncthreads();
    }

    dist[blockIdx.y*GPU_TILE_DIM*n+blockIdx.x*GPU_TILE_DIM + row*n + col] = a[row*GPU_TILE_DIM + col]; //Tij
    __syncthreads();
}
/*
 *  FILLME: Use different kernels for the different stages of the
 *  tiled FW computation
 *  
 *  Use GPU_TILE_DIM (see graph.h) as the tile dimension. You can
 *  adjust its value during compilation. See `make help' for more
 *  information.
 */ 

graph_t *MAKE_KERNEL_NAME(_gpu, _naive)(graph_t *graph)
{
    xtimer_t transfer_timer;
    timer_clear(&transfer_timer);
    timer_start(&transfer_timer);
    weight_t *dist_gpu = copy_graph_to_gpu(graph);
    timer_stop(&transfer_timer);
    
    //init block and grid
    dim3 block(8,8);
    dim3 grid((graph->nr_vertices*graph->nr_vertices)/64); // this should change

    //call the GPU kernel
    for(int k=0;k<graph->nr_vertices;k++) { //main loop
        GPU_KERNEL_NAME(_naive)<<<grid, block>>>(dist_gpu,graph->nr_vertices,k);
        hipDeviceSynchronize();
    }

     /* Wait for last kernel to finish, so as to measure correctly the
     * transfer times Otherwise, copy from GPU will block
     */
    hipDeviceSynchronize();

    /* Copy back results to host */
    timer_start(&transfer_timer);
    copy_graph_from_gpu(dist_gpu, graph);
    timer_stop(&transfer_timer);
    printf("Total transfer times: %lf s\n",
           timer_elapsed_time(&transfer_timer));
    return graph;
}

graph_t *MAKE_KERNEL_NAME(_gpu, _tiled)(graph_t *graph)
{
    xtimer_t transfer_timer;
    timer_clear(&transfer_timer);
    timer_start(&transfer_timer);
    weight_t *dist_gpu = copy_graph_to_gpu(graph);
    timer_stop(&transfer_timer);

    int tile_no = graph->nr_vertices / GPU_TILE_DIM;
    for (int k=0;k< tile_no;k++) { // k = K from the paper, be careful with sizes

        //phase one
        dim3 block(GPU_TILE_DIM, GPU_TILE_DIM);
        dim3 grid1(1);
        GPU_KERNEL_NAME(_tiled_stage_1)<<<grid1, block>>>(dist_gpu,graph->nr_vertices,k);
             
        //phase two
        dim3 grid2(1,tile_no);
        GPU_KERNEL_NAME(_tiled_stage_2)<<<grid2, block>>>(dist_gpu,graph->nr_vertices,k);       
        
        //phase three    
        dim3 grid3(tile_no);
        GPU_KERNEL_NAME(_tiled_stage_3)<<<grid3, block>>>(dist_gpu,graph->nr_vertices,k);

        dim3 grid4(tile_no,tile_no);
        GPU_KERNEL_NAME(_tiled_stage_4)<<<grid4, block>>>(dist_gpu,graph->nr_vertices,k);
    }
    /*
     * Wait for last kernel to finish, so as to measure correctly the
     * transfer times Otherwise, copy from GPU will block
     */
    hipDeviceSynchronize();

    /* Copy back results to host */
    timer_start(&transfer_timer);
    copy_graph_from_gpu(dist_gpu, graph);
    timer_stop(&transfer_timer);
    printf("Total transfer times: %lf s\n",
            timer_elapsed_time(&transfer_timer));
    return graph;
}

graph_t *MAKE_KERNEL_NAME(_gpu, _tiled_shmem)(graph_t *graph)
{
    xtimer_t transfer_timer;
    timer_clear(&transfer_timer);
    timer_start(&transfer_timer);
    weight_t *dist_gpu = copy_graph_to_gpu(graph);
    timer_stop(&transfer_timer);

    int tile_no = graph->nr_vertices / GPU_TILE_DIM;
    for (int k=0;k< tile_no;k++) { // k = K from the paper, be careful with sizes

        //phase one
        dim3 block(GPU_TILE_DIM, GPU_TILE_DIM);
        dim3 grid1(1);
        GPU_KERNEL_NAME(_tiled_shmem_stage_1)<<<grid1, block>>>(dist_gpu,graph->nr_vertices,k);        

        //phase two
        dim3 grid2(1,tile_no);
        GPU_KERNEL_NAME(_tiled_shmem_stage_2)<<<grid2, block>>>(dist_gpu,graph->nr_vertices,k);        
        
        //phase three    
        dim3 grid3(tile_no);
        GPU_KERNEL_NAME(_tiled_shmem_stage_3)<<<grid3, block>>>(dist_gpu,graph->nr_vertices,k);

        dim3 grid4(tile_no,tile_no);
        GPU_KERNEL_NAME(_tiled_shmem_stage_4)<<<grid4, block>>>(dist_gpu,graph->nr_vertices,k);
    }

    /*
     * FILLME: Set up and launch the kernel(s)
     *
     * You may need different grid/block configurations for each stage
     * of the computation
     * 
     * Use GPU_TILE_DIM (see graph.h) as the tile dimension. You can
     * adjust its value during compilation. See `make help' for more
     * information.
     */

    /*
     * Wait for last kernel to finish, so as to measure correctly the
     * transfer times Otherwise, copy from GPU will block
     */
    hipDeviceSynchronize();

    /* Copy back results to host */
    timer_start(&transfer_timer);
    copy_graph_from_gpu(dist_gpu, graph);
    timer_stop(&transfer_timer);
    printf("Total transfer times: %lf s\n",
           timer_elapsed_time(&transfer_timer));
    return graph;
            
}
