#include "hip/hip_runtime.h"
/*
 *  gpu_kernels.cu -- GPU kernels
 *
 *  Copyright (C) 2014, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2014, Vasileios Karakasis
 */ 

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "error.h"
#include "gpu_util.h"
#include "graph.h"
#include "timer.h"

#define GPU_KERNEL_NAME(name)   do_apsp_gpu ## name

weight_t *copy_graph_to_gpu(const graph_t *graph)
{
    size_t dist_size = graph->nr_vertices*graph->nr_vertices;
    weight_t *dist_gpu = (weight_t *) gpu_alloc(dist_size*sizeof(*dist_gpu));
    if (!dist_gpu)
        error(0, "gpu_alloc() failed: %s", gpu_get_last_errmsg());

    if (copy_to_gpu(graph->weights[0], dist_gpu,
                    dist_size*sizeof(*dist_gpu)) < 0)
        error(0, "copy_to_gpu() failed: %s", gpu_get_last_errmsg());

    return dist_gpu;
}

graph_t *copy_graph_from_gpu(const weight_t *dist_gpu, graph_t *graph)
{
    size_t dist_size = graph->nr_vertices*graph->nr_vertices;

    if (copy_from_gpu(graph->weights[0], dist_gpu,
                      dist_size*sizeof(*dist_gpu)) < 0)
        error(0, "copy_from_gpu() failed: %s", gpu_get_last_errmsg());

    return graph;
}

/*
 *  The naive GPU kernel
 */ 
__global__ void GPU_KERNEL_NAME(_naive)(weight_t *dist, int n, int k)
{
    // FILLME: the naive GPU kernel code
}

/*
 *  The tiled GPU kernel(s) using global memory
 */ 
__global__ void GPU_KERNEL_NAME(_tiled_stage_X)(weight_t *dist, int n,
                                                int k_tile)
{
    // FILLME: tiled GPU kernel code for stage X
}

/*
 *  FILLME: Use different kernels for the different stages of the
 *  tiled FW computation
 *
 *  Use GPU_TILE_DIM (see graph.h) as the tile dimension. You can
 *  adjust its value during compilation. See `make help' for more
 *  information.
 */ 

/*
 *  The tiled GPU kernel(s) using shared memory
 */ 
__global__ void GPU_KERNEL_NAME(_tiled_shmem_stage_X)(weight_t *dist, int n,
                                                      int k_tile)
{
    // FILLME: tiled GPU kernel code using shared memory for stage X
}

/*
 *  FILLME: Use different kernels for the different stages of the
 *  tiled FW computation
 *  
 *  Use GPU_TILE_DIM (see graph.h) as the tile dimension. You can
 *  adjust its value during compilation. See `make help' for more
 *  information.
 */ 

graph_t *MAKE_KERNEL_NAME(_gpu, _naive)(graph_t *graph)
{
    xtimer_t transfer_timer;
    timer_clear(&transfer_timer);
    timer_start(&transfer_timer);
    weight_t *dist_gpu = copy_graph_to_gpu(graph);
    timer_stop(&transfer_timer);

    /* FILLME: Set up and launch the kernel(s) */
    
    /*
     * Wait for last kernel to finish, so as to measure correctly the
     * transfer times Otherwise, copy from GPU will block
     */
    hipDeviceSynchronize();

    /* Copy back results to host */
    timer_start(&transfer_timer);
    copy_graph_from_gpu(dist_gpu, graph);
    timer_stop(&transfer_timer);
    printf("Total transfer times: %lf s\n",
           timer_elapsed_time(&transfer_timer));
    return graph;
}

graph_t *MAKE_KERNEL_NAME(_gpu, _tiled)(graph_t *graph)
{
    xtimer_t transfer_timer;
    timer_clear(&transfer_timer);
    timer_start(&transfer_timer);
    weight_t *dist_gpu = copy_graph_to_gpu(graph);
    timer_stop(&transfer_timer);

    /*
     * FILLME: Set up and launch the kernel(s)
     *
     * You may need different grid/block configurations for each stage
     * of the computation
     * 
     * Use GPU_TILE_DIM (see graph.h) as the tile dimension. You can
     * adjust its value during compilation. See `make help' for more
     * information.
     */

    /*
     * Wait for last kernel to finish, so as to measure correctly the
     * transfer times Otherwise, copy from GPU will block
     */
    hipDeviceSynchronize();

    /* Copy back results to host */
    timer_start(&transfer_timer);
    copy_graph_from_gpu(dist_gpu, graph);
    timer_stop(&transfer_timer);
    printf("Total transfer times: %lf s\n",
           timer_elapsed_time(&transfer_timer));
    return graph;
}

graph_t *MAKE_KERNEL_NAME(_gpu, _tiled_shmem)(graph_t *graph)
{
    xtimer_t transfer_timer;
    timer_clear(&transfer_timer);
    timer_start(&transfer_timer);
    weight_t *dist_gpu = copy_graph_to_gpu(graph);
    timer_stop(&transfer_timer);

    /*
     * FILLME: Set up and launch the kernel(s)
     *
     * You may need different grid/block configurations for each stage
     * of the computation
     * 
     * Use GPU_TILE_DIM (see graph.h) as the tile dimension. You can
     * adjust its value during compilation. See `make help' for more
     * information.
     */

    /*
     * Wait for last kernel to finish, so as to measure correctly the
     * transfer times Otherwise, copy from GPU will block
     */
    hipDeviceSynchronize();

    /* Copy back results to host */
    timer_start(&transfer_timer);
    copy_graph_from_gpu(dist_gpu, graph);
    timer_stop(&transfer_timer);
    printf("Total transfer times: %lf s\n",
           timer_elapsed_time(&transfer_timer));
    return graph;
}
