#include "hip/hip_runtime.h"
/*
 *  gpu_kernels.cu -- GPU kernels
 *
 *  Copyright (C) 2014, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2014, Vasileios Karakasis
 */ 

#include <stdio.h>
#include <hip/hip_runtime.h>
#include "error.h"
#include "gpu_util.h"
#include "graph.h"
#include "timer.h"

#define GPU_KERNEL_NAME(name)   do_apsp_gpu ## name

weight_t *copy_graph_to_gpu(const graph_t *graph)
{
    size_t dist_size = graph->nr_vertices*graph->nr_vertices;
    weight_t *dist_gpu = (weight_t *) gpu_alloc(dist_size*sizeof(*dist_gpu));
    if (!dist_gpu)
        error(0, "gpu_alloc() failed: %s", gpu_get_last_errmsg());

    if (copy_to_gpu(graph->weights[0], dist_gpu,
                    dist_size*sizeof(*dist_gpu)) < 0)
        error(0, "copy_to_gpu() failed: %s", gpu_get_last_errmsg());

    return dist_gpu;
}

graph_t *copy_graph_from_gpu(const weight_t *dist_gpu, graph_t *graph)
{
    size_t dist_size = graph->nr_vertices*graph->nr_vertices;

    if (copy_from_gpu(graph->weights[0], dist_gpu,
                      dist_size*sizeof(*dist_gpu)) < 0)
        error(0, "copy_from_gpu() failed: %s", gpu_get_last_errmsg());

    return graph;
}

/*
 *  The naive GPU kernel
 */ 
__global__ void GPU_KERNEL_NAME(_naive)(weight_t *dist, int n, int k)
{
    int tid = (blockDim.x*blockDim.y*blockIdx.x)+ // consider line grid
        (threadIdx.x*blockDim.y + threadIdx.y);   // and global order 

    if (tid > n*n)
        return;

    int row = tid / n;
    int col = tid % n;

    dist[tid] = MIN(dist[tid], dist[row*n+k]+dist[k*n+col]);  
}

/*
 *  The tiled GPU kernel(s) using global memory
 */ 
__global__ void GPU_KERNEL_NAME(_tiled_stage_X)(weight_t *dist, int n,
        int k_tile, int kk)
{
    int tid = threadIdx.x*blockDim.y + threadIdx.y;  
    
    weight_t * a;
    weight_t * b;
    weight_t * c;
    
    /*case single block or column grid*/
    if (gridDim.x == 1) {

        /*signle block*/
        if (gridDim.y == 1) {
            a = &dist[k_tile*GPU_TILE_DIM+k_tile*GPU_TILE_DIM*n]; // Tkk
            b = a;
            c = a;
        } else { /*column grid*/
            if (blockIdx.y == k_tile)
                return;
            a = &dist[k_tile*GPU_TILE_DIM+blockIdx.y*GPU_TILE_DIM*n]; // Tik
            b = a;
            c = &dist[k_tile*GPU_TILE_DIM+k_tile*GPU_TILE_DIM*n]; //Tkk
        }

    } else { /*case column grid or square grid*/
        
        if (gridDim.y == 1) { /*line grid*/
            if (blockIdx.x == k_tile)
                return;
            a = &dist[k_tile*GPU_TILE_DIM*n+blockIdx.x*GPU_TILE_DIM]; //Tki
            b = &dist[k_tile*GPU_TILE_DIM+k_tile*GPU_TILE_DIM*n]; //Tkk
            c = a;
        } else { /*square grid*/
            if ((blockIdx.x == k_tile) || (blockIdx.y == k_tile))
                return;
            a = &dist[blockIdx.y*GPU_TILE_DIM*n+blockIdx.x*GPU_TILE_DIM]; //Tij
            b = &dist[blockIdx.y*n*GPU_TILE_DIM+k_tile*GPU_TILE_DIM]; //Tik
            c = &dist[k_tile*GPU_TILE_DIM*n+blockIdx.x*GPU_TILE_DIM]; //Tkj
        }

    }
    
    int row = tid / GPU_TILE_DIM;
    int col = tid % GPU_TILE_DIM;

    a[tid] = MIN(a[tid], b[row*n+kk]+c[kk*n+col]);  
}

/*
 *  FILLME: Use different kernels for the different stages of the
 *  tiled FW computation
 *
 *  Use GPU_TILE_DIM (see graph.h) as the tile dimension. You can
 *  adjust its value during compilation. See `make help' for more
 *  information.
 */ 

/*
 *  The tiled GPU kernel(s) using shared memory
 */ 
__global__ void GPU_KERNEL_NAME(_tiled_shmem_stage_X)(weight_t *dist, int n,
                                                      int k_tile)
{
    // FILLME: tiled GPU kernel code using shared memory for stage X
}

/*
 *  FILLME: Use different kernels for the different stages of the
 *  tiled FW computation
 *  
 *  Use GPU_TILE_DIM (see graph.h) as the tile dimension. You can
 *  adjust its value during compilation. See `make help' for more
 *  information.
 */ 

graph_t *MAKE_KERNEL_NAME(_gpu, _naive)(graph_t *graph)
{
    xtimer_t transfer_timer;
    timer_clear(&transfer_timer);
    timer_start(&transfer_timer);
    weight_t *dist_gpu = copy_graph_to_gpu(graph);
    timer_stop(&transfer_timer);
    
    //init block and grid
    dim3 block(8,8);
    dim3 grid((graph->nr_vertices*graph->nr_vertices)/64); // this should change

    //call the GPU kernel
    for(int k=0;k<graph->nr_vertices;k++) { //main loop
        GPU_KERNEL_NAME(_naive)<<<grid, block>>>(dist_gpu,graph->nr_vertices,k);
        hipDeviceSynchronize();
    }

     /* Wait for last kernel to finish, so as to measure correctly the
     * transfer times Otherwise, copy from GPU will block
     */
    hipDeviceSynchronize();

    /* Copy back results to host */
    timer_start(&transfer_timer);
    copy_graph_from_gpu(dist_gpu, graph);
    timer_stop(&transfer_timer);
    printf("Total transfer times: %lf s\n",
           timer_elapsed_time(&transfer_timer));
    return graph;
}

graph_t *MAKE_KERNEL_NAME(_gpu, _tiled)(graph_t *graph)
{
    xtimer_t transfer_timer;
    timer_clear(&transfer_timer);
    timer_start(&transfer_timer);
    weight_t *dist_gpu = copy_graph_to_gpu(graph);
    timer_stop(&transfer_timer);

    int tile_no = graph->nr_vertices / GPU_TILE_DIM;

    for (int kk=0;kk< tile_no;kk++) { // kk = K from the paper, be careful with sizes
    
        //phase one
        dim3 block(GPU_TILE_DIM, GPU_TILE_DIM);
        dim3 grid1(1);
        for(int k=0;k<GPU_TILE_DIM;k++) {
            GPU_KERNEL_NAME(_tiled_shmem)<<<grid1, block>>>(dist_gpu,graph->nr_vertices,k,kk);
            hipDeviceSynchronize();
        }

        dim3 grid2(tile_no);
        dim3 grid3(1,tile_no);
        for(int k=0;k<GPU_TILE_DIM;k++) {
            GPU_KERNEL_NAME(_tiled_shmem)<<<grid2, block>>>(dist_gpu,graph->nr_vertices,k,kk);
            hipDeviceSynchronize();
            GPU_KERNEL_NAME(_tiled_shmem)<<<grid3, block>>>(dist_gpu,graph->nr_vertices,k,kk);
            hipDeviceSynchronize();
        }

        dim3 grid4(tile_no,tile_no);
        for(int k=0;k<GPU_TILE_DIM;k++) {
            GPU_KERNEL_NAME(_tiled_shmem)<<<grid4, block>>>(dist_gpu,graph->nr_vertices,k,kk);
            hipDeviceSynchronize();
        }

    }
    /*
     * Wait for last kernel to finish, so as to measure correctly the
     * transfer times Otherwise, copy from GPU will block
     */
    hipDeviceSynchronize();

    /* Copy back results to host */
    timer_start(&transfer_timer);
    copy_graph_from_gpu(dist_gpu, graph);
    timer_stop(&transfer_timer);
    printf("Total transfer times: %lf s\n",
           timer_elapsed_time(&transfer_timer));
    return graph;
}

graph_t *MAKE_KERNEL_NAME(_gpu, _tiled_shmem)(graph_t *graph)
{
    xtimer_t transfer_timer;
    timer_clear(&transfer_timer);
    timer_start(&transfer_timer);
    weight_t *dist_gpu = copy_graph_to_gpu(graph);
    timer_stop(&transfer_timer);

    /*
     * FILLME: Set up and launch the kernel(s)
     *
     * You may need different grid/block configurations for each stage
     * of the computation
     * 
     * Use GPU_TILE_DIM (see graph.h) as the tile dimension. You can
     * adjust its value during compilation. See `make help' for more
     * information.
     */

    /*
     * Wait for last kernel to finish, so as to measure correctly the
     * transfer times Otherwise, copy from GPU will block
     */
    hipDeviceSynchronize();

    /* Copy back results to host */
    timer_start(&transfer_timer);
    copy_graph_from_gpu(dist_gpu, graph);
    timer_stop(&transfer_timer);
    printf("Total transfer times: %lf s\n",
           timer_elapsed_time(&transfer_timer));
    return graph;
            
}
